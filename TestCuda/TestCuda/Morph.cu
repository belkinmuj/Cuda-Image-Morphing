#include "hip/hip_runtime.h"
#include "Morph.cuh"
#include "Image.cuh"

DeviceMorph::DeviceMorph(const cimg_library::CImg<unsigned char>& imageSrc, const cimg_library::CImg<unsigned char>& imageDest, const std::vector<Point>& pointsSrc, const std::vector<Point>& pointsDest, const std::vector<IndexTriangle>& triangles)
{
	if (!(imageSrc.width() == imageDest.width() &&
		imageSrc.height() == imageDest.height() &&
		imageSrc.depth() == imageDest.depth() &&
		imageSrc.spectrum() == imageDest.spectrum()))
	{
		throw std::invalid_argument("Image source must be same width/height/depth/spectrum as destination.");
	}

	for (size_t triangleIndex = 0; triangleIndex < triangles.size(); triangleIndex++)
	{
		const IndexTriangle& triangle = triangles[triangleIndex];
		for (size_t pIndex = 0; pIndex < 3; pIndex ++)
		{
			if (!(triangle.points[pIndex] < pointsSrc.size() && triangle.points[pIndex] < pointsDest.size()))
			{
				throw std::invalid_argument("Invalid triangulation for the given points.");
			}
		}
	}

	_imageSrcTexture = new CImgTexture(imageSrc);
	hipMalloc(&d_imageSrcTexture, sizeof(CImgTexture));
	hipMemcpy(d_imageSrcTexture, _imageSrcTexture, sizeof(CImgTexture), hipMemcpyHostToDevice);

	_imageDestTexture = new CImgTexture(imageDest);
	hipMalloc(&d_imageDestTexture, sizeof(CImgTexture));
	hipMemcpy(d_imageDestTexture, _imageDestTexture, sizeof(CImgTexture), hipMemcpyHostToDevice);

	_output = new DeviceImage(imageSrc);
	hipMalloc(&d_output, sizeof(DeviceImage));
	hipMemcpy(d_output, _output, sizeof(DeviceImage), hipMemcpyHostToDevice);

	const Point* pointsSrcData = pointsSrc.data();
	hipMalloc(&d_pointsSrc, sizeof(Point) * pointsSrc.size());
	hipMemcpy(d_pointsSrc, pointsSrcData, sizeof(Point) * pointsSrc.size(), hipMemcpyHostToDevice);

	const Point* pointsDestData = pointsDest.data();
	hipMalloc(&d_pointsDest, sizeof(Point) * pointsDest.size());
	hipMemcpy(d_pointsDest, pointsDestData, sizeof(Point) * pointsDest.size(), hipMemcpyHostToDevice);

	const IndexTriangle* trianglesData = triangles.data();
	hipMalloc(&d_triangles, sizeof(IndexTriangle) * triangles.size());
	hipMemcpy(d_triangles, trianglesData, sizeof(IndexTriangle) * triangles.size(), hipMemcpyHostToDevice);

	_trianglesSize = triangles.size();

	hipMalloc(&d_instance, sizeof(DeviceMorph));
	hipMemcpy(d_instance, this, sizeof(DeviceMorph), hipMemcpyHostToDevice);
}

DeviceMorph::~DeviceMorph()
{
	hipFree(d_pointsSrc);
	hipFree(d_pointsDest);
	hipFree(d_triangles);
	hipFree(d_instance);

	delete _imageSrcTexture;
	delete _imageDestTexture;

	delete _output;
}

__host__ __device__ 
Point computePosition(Point& p, const Point* pointsSrc, const Point* pointsDest, const IndexTriangle* triangles, const size_t& trianglesSize, const double& ratio = 1)
{
	for (size_t trIdx = 0; trIdx < trianglesSize; trIdx++)
	{
		const Point& p1 = pointsDest[triangles[trIdx].points[0]];
		const Point& p2 = pointsDest[triangles[trIdx].points[1]];
		const Point& p3 = pointsDest[triangles[trIdx].points[2]];

		double bot = (p2.y - p3.y) * (p1.x - p3.x) + (p3.x - p2.x) * (p1.y - p3.y);
		double sTop = (p2.y - p3.y) * (p.x - p3.x) + (p3.x - p2.x) * (p.y - p3.y);
		double tTop = (p3.y - p1.y) * (p.x - p3.x) + (p1.x - p3.x) * (p.y - p3.y);

		double s = sTop / bot;
		double t = tTop / bot;

		if (!(s >= 0 && s <= 1 && t >= 0 && t <= 1 && (s + t) <= 1))
		{
			continue;
		}

		const Point& destp0 = pointsSrc[triangles[trIdx].points[0]];
		const Point& destp1 = pointsSrc[triangles[trIdx].points[1]];
		const Point& destp2 = pointsSrc[triangles[trIdx].points[2]];

		Point destp;
		destp.x = s * destp0.x + t * destp1.x + (1 - s - t) * destp2.x;
		destp.y = s * destp0.y + t * destp1.y + (1 - s - t) * destp2.y;

		destp.x = destp.x * ratio + p.x * (1 - ratio);
		destp.y = destp.y * ratio + p.y * (1 - ratio);

		return destp;
	}
}

__global__ 
void morphKernel(DeviceMorph* d_instance, double ratio)
{
	Point p;
	p.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	p.y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (!(p.x >= 0 && p.x < d_instance->d_output->width() && p.y >= 0 && p.y < d_instance->d_output->height()))
	{
		return;
	}

	Point srcPoint = computePosition(p, d_instance->d_pointsSrc, d_instance->d_pointsDest, d_instance->d_triangles, d_instance->_trianglesSize, ratio);
	Point destPoint = computePosition(p, d_instance->d_pointsDest, d_instance->d_pointsSrc, d_instance->d_triangles, d_instance->_trianglesSize, 1 - ratio);

	uchar4 srcPixel = d_instance->d_imageSrcTexture->cubicTex2D(srcPoint.x, srcPoint.y);
	uchar4 destPixel = d_instance->d_imageDestTexture->cubicTex2D(destPoint.x, destPoint.y);

	d_instance->d_output->at(p.x, p.y, 0, 0) = srcPixel.x * (1 - ratio) + destPixel.x * ratio;
	d_instance->d_output->at(p.x, p.y, 0, 1) = srcPixel.y * (1 - ratio) + destPixel.y * ratio;
	d_instance->d_output->at(p.x, p.y, 0, 2) = srcPixel.z * (1 - ratio) + destPixel.z * ratio;
}

__global__
void warpKernel(DeviceMorph* d_instance, double ratio, int way)
{
	Point p;
	p.x = (blockIdx.x * blockDim.x) + threadIdx.x;
	p.y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (!(p.x >= 0 && p.x < d_instance->d_output->width() && p.y >= 0 && p.y < d_instance->d_output->height()))
	{
		return;
	}

	if (way == 1)
	{
		Point srcPoint = computePosition(p, d_instance->d_pointsSrc, d_instance->d_pointsDest, d_instance->d_triangles, d_instance->_trianglesSize, ratio);
		uchar4 srcPixel = tex2D<uchar4>(d_instance->d_imageDestTexture->tex, srcPoint.x + 0.5f, srcPoint.y + 0.5f);
		d_instance->d_output->at(p.x, p.y, 0, 0) = srcPixel.x;
		d_instance->d_output->at(p.x, p.y, 0, 1) = srcPixel.y;
		d_instance->d_output->at(p.x, p.y, 0, 2) = srcPixel.z;
	}
	else if (way == 2)
	{
		Point destPoint = computePosition(p, d_instance->d_pointsDest, d_instance->d_pointsSrc, d_instance->d_triangles, d_instance->_trianglesSize, ratio);
		uchar4 destPixel = tex2D<uchar4>(d_instance->d_imageSrcTexture->tex, destPoint.x + 0.5f, destPoint.y + 0.5f);
		d_instance->d_output->at(p.x, p.y, 0, 0) = destPixel.x;
		d_instance->d_output->at(p.x, p.y, 0, 1) = destPixel.y;
		d_instance->d_output->at(p.x, p.y, 0, 2) = destPixel.z;
	}
}

std::vector<cimg_library::CImg<unsigned char>> DeviceMorph::computeMorph(const size_t threadsX, const size_t threadsY) const
{
	int size = _output->size();
	cimg_library::CImg<unsigned char> cImg(_output->width(), _output->height(), _output->depth(), _output->spectrum());
	std::vector<cimg_library::CImg<unsigned char>> frames;

	dim3 threadsPerBlock(threadsX, threadsY);
	dim3 numBlocks((_output->width() / threadsPerBlock.x) + 1, (_output->height() / threadsPerBlock.y) + 1);

	double step = 0.02;
	for (double r = step; r <= 1.0; r += step)
	{
		morphKernel<<< numBlocks, threadsPerBlock >>>(d_instance, r);
		hipMemcpy(cImg._data, _output->data(), sizeof(unsigned char) * size, hipMemcpyDeviceToHost);
		frames.push_back(cImg);
		printf("Done with frame step %.3f\n", r);
	}

	return frames;
}


cimg_library::CImg<unsigned char> DeviceMorph::computeWarp(double ratio, int way, const size_t threadsX, const size_t threadsY) const
{
	int size = _output->size();

	cimg_library::CImg<unsigned char> cImg(_output->width(), _output->height(), _output->depth(), _output->spectrum());
	dim3 threadsPerBlock(threadsX, threadsY);
	dim3 numBlocks((_output->width() / threadsPerBlock.x) + 1, (_output->height() / threadsPerBlock.y) + 1);

	warpKernel<<< numBlocks, threadsPerBlock >>>(d_instance, ratio, way);
	hipMemcpy(cImg._data, _output->data(), sizeof(unsigned char) * size, hipMemcpyDeviceToHost);

	return cImg;
}

